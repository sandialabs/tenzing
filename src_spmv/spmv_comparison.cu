#include "hip/hip_runtime.h"
/* use MCTS on a particular assignment of operations to streams

   yl is in stream1, other GPU operations in stream2
*/

#include "sched/hip/hip_runtime.h"
#include "sched/schedule.hpp"
#include "sched/graph.hpp"
#include "sched/numeric.hpp"
#include "sched/mcts.hpp"
#include "sched/ops_cuda.hpp"

#include "ops_spmv.cuh"

#include "where.hpp"
#include "csr_mat.hpp"
#include "row_part_spmv.cuh"

#include "mm/mm.hpp"

#include <algorithm>
#include <numeric>
#include <chrono>
#include <thread>
#include <iostream>

#include <hipsparse.h>

typedef int Ordinal;
typedef float Scalar;
typedef MtxReader<Ordinal, Scalar> reader_t;
typedef typename reader_t::coo_type mm_coo_t;
typedef typename reader_t::csr_type mm_csr_t;

template <Where w>
using csr_type = CsrMat<w, Ordinal, Scalar>;

template <typename Benchmarker>
int do_comparison(Benchmarker &benchmarker, const std::string &matrixPath) {
    int rank = 0;
    int size = 1;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    MPI_Barrier(MPI_COMM_WORLD);

    {

        char hostname[MPI_MAX_PROCESSOR_NAME] = {};
        int len;
        MPI_Get_processor_name(hostname, &len);

        const char *p = std::getenv("OMP_PLACES");
        if (!p) p = "<unset>";
        std::cerr << "rank " << rank << " on " << hostname << " OMP_PLACES: " << p << "\n";

        // round-robin GPU scheduling
        int devcount;
        CUDA_RUNTIME(hipGetDeviceCount(&devcount));
        int dev = rank % devcount;
        CUDA_RUNTIME(hipSetDevice(dev));

        hipDeviceProp_t prop;
        CUDA_RUNTIME(hipGetDeviceProperties(&prop, dev));
        fprintf(stderr, "rank %d on %s GPU %08x:%02x:%02x.0 (%d)\n", rank, hostname, prop.pciDomainID, prop.pciBusID, prop.pciDeviceID, dev);

    }

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    /* ensure streams are numerically ordered, so that later when ranks sort by stream,
       stream1 is the smallest on both ranks
    */
    if (stream1 > stream2) std::swap(stream1, stream2);



    /* interesting parameters:
       vortex: 1.5m rows, 15mnnz, bandwidth = 15m/16 4 nodes, 4 ranks per node
       may be even more interesting with 12 nodes, 4 ranks per node
    */

    int m = 150000;
    int bw = m / size;
    int nnz = m * 10;

    csr_type<Where::host> A;


    if ("" == matrixPath) {

        // generate and distribute A
        if (0 == rank)
        {
            std::cerr << "generate matrix\n";
            A = random_band_matrix<Ordinal, Scalar>(m, bw, nnz);
        }
    } else {
        if (0 == rank) {
            std::cerr << "load " << matrixPath << std::endl;
            reader_t reader(matrixPath);
            mm_coo_t coo = reader.read_coo();
            mm_csr_t csr(coo);

            std::cerr << "allocate A\n";
            A = csr_type<Where::host>(csr.num_rows(), csr.num_cols(), csr.nnz());

            std::cerr << "move CSR data...\n";
            for (size_t i = 0; i < csr.row_ptr().size(); ++i) {
                A.row_ptr()[i] = csr.row_ptr(i);
            }
            for (size_t i = 0; i < csr.col_ind().size(); ++i) {
                A.col_ind()[i] = csr.col_ind(i);
            }
            for (size_t i = 0; i < csr.val().size(); ++i) {
                A.val()[i] = csr.val(i);
            }
        }
    }

    RowPartSpmv<Ordinal, Scalar> spmv(A, 0, MPI_COMM_WORLD);

    std::shared_ptr<Start> start = std::make_shared<Start>();

    std::shared_ptr<StreamedOp> scatter;
    {
        Scatter::Args args{
            .dst = spmv.x_send_buf().view(),
            .src = spmv.lx().view(),
            .idx = spmv.x_send_idx().view()};
        auto _scatter = std::make_shared<Scatter>(args);
        scatter = std::make_shared<StreamedOp>(_scatter, stream2);
    }

    std::shared_ptr<StreamedOp> yl, yr;
    {
        SpMV<Ordinal, Scalar>::Args rArgs, lArgs;
        rArgs.a = spmv.rA().view();
        rArgs.y = spmv.ly().view(); // FIXME: remote y?
        rArgs.x = spmv.rx().view();
        lArgs.a = spmv.lA().view();
        lArgs.y = spmv.ly().view();
        lArgs.x = spmv.lx().view();
        auto _yl = std::make_shared<SpMV<Ordinal, Scalar>>("yl", lArgs);
        auto _yr = std::make_shared<SpMV<Ordinal, Scalar>>("yr", rArgs);

        // yl and yr in different streams
        yl = std::make_shared<StreamedOp>(_yl, stream1);
        yr = std::make_shared<StreamedOp>(_yr, stream2);
    }

    std::shared_ptr<PostSend> postSend;
    std::shared_ptr<WaitSend> waitSend;
    {
        PostSend::Args args;
        for (auto &arg : spmv.send_params())
        {
            if (arg.displ + arg.count > spmv.x_send_buf().size()) throw std::logic_error(AT);
            if (!spmv.x_send_buf().data()) throw std::logic_error(AT);
            args.sends.push_back(Isend::Args{
                .buf = spmv.x_send_buf().data() + arg.displ,
                .count = arg.count,
                .datatype = MPI_FLOAT,
                .dest = arg.dst,
                .tag = 0,
                .comm = MPI_COMM_WORLD,
                .request = &arg.req});
        }
        postSend = std::make_shared<PostSend>(args);
        waitSend = std::make_shared<WaitSend>(args);
    }

    std::shared_ptr<PostRecv> postRecv;
    std::shared_ptr<WaitRecv> waitRecv;
    {
        PostRecv::Args args;
        for (auto &arg : spmv.recv_params())
        {
            if (arg.displ + arg.count > spmv.rx().size()) throw std::logic_error(AT);
            if (!spmv.rx().data()) throw std::logic_error(AT);
            args.recvs.push_back(Irecv::Args{
                .buf = spmv.rx().data() + arg.displ,
                .count = arg.count,
                .datatype = MPI_FLOAT,
                .source = arg.src,
                .tag = 0,
                .comm = MPI_COMM_WORLD,
                .request = &arg.req});
        }
        postRecv = std::make_shared<PostRecv>(args);
        waitRecv = std::make_shared<WaitRecv>(args);
    }
    std::shared_ptr<StreamedOp> y;
    {
        VectorAdd::Args args;
        auto _y = std::make_shared<VectorAdd>("y", args);
        y = std::make_shared<StreamedOp>(_y, stream2);
    }
    std::shared_ptr<End> end = std::make_shared<End>();

    std::cerr << "create graph\n";
    Graph<Node> orig(start);

    // immediately recv, local spmv, or scatter
    orig.then(start, yl);
    orig.then(start, postRecv);
    orig.then(orig.then(start, scatter), postSend);

    // remote matrix after recv
    orig.then(waitRecv, yr);

    // add after local and remote done, then end
    orig.then(yl, y);
    orig.then(yr, y);

    // end once add and send is done
    orig.then(y, end);
    orig.then(waitSend, end);

    // initiate sends and recvs before waiting for either
    orig.then(postSend, waitSend);
    orig.then(postSend, waitRecv);
    orig.then(postRecv, waitSend);
    orig.then(postRecv, waitRecv);

    orig.dump();
    MPI_Barrier(MPI_COMM_WORLD);

    std::vector<Graph<Node>> gpuGraphs;
    gpuGraphs.push_back(orig);

    if (0 == rank) {
        std::cerr << gpuGraphs.size() << " GpuNode graphs\n";
    }

    if (0 == rank) {
#if 1
        for (auto &graph : gpuGraphs) {
            graph.dump();
            std::cerr << "\n";
        }
#endif
#if 0
        gpuGraphs.begin()->dump();
        std::cerr << "\n";
        (--gpuGraphs.end())->dump();
#endif
    }

    MPI_Barrier(MPI_COMM_WORLD);

    
    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "insert sync...\n";
    std::vector<Graph<Node>> syncedGraphs;
    for (auto &graph : gpuGraphs) {
        auto next = insert_synchronization(graph);
        syncedGraphs.push_back(next);
    }

    if (0 == rank) {
        std::cerr << "created " << syncedGraphs.size() << " sync graphs:\n";
    }


    if (0 == rank) {
        syncedGraphs.begin()->dump();
        std::cerr << "\n";
    }



    if (0 == rank) {
        for (auto &graph : syncedGraphs) {
            graph.dump();
            std::cerr << "\n";
        }
    }


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "convert to cpu graphs...\n";
    std::vector<Graph<CpuNode>> cpuGraphs;
    for (auto &graph : syncedGraphs) {
        cpuGraphs.push_back(graph.nodes_cast<CpuNode>());
    }
    if (0 == rank) std::cerr << "converted " << cpuGraphs.size() << " graphs\n";

    std::shared_ptr<CpuNode> streamWait;
    std::shared_ptr<CpuNode> cer1;
    std::shared_ptr<CpuNode> ces1;
    std::shared_ptr<CpuNode> cer2;
    std::shared_ptr<CpuNode> ces2;

    // find the added sync nodes
    for (const auto &kv : cpuGraphs[0].succs_) {
        const auto &op = kv.first;
        if ( "StreamWait-after-yl-b4-y" == op->name()) {
            streamWait = op;
        } else if ("CudaEventRecord-after-Scatter-b4-PostSend" == op->name()) {
            cer1 = op;
        } else if ("CudaEventSync-after-b4-PostSend" == op->name()) {
            ces1 = op;
        } else if ("CudaEventRecord-after-y-b4-end" == op->name()) {
            cer2 = op;
        } else if ("CudaEventSync-after-y-b4-end" == op->name()) {
            ces2 = op;
        }
    }

    if (!streamWait) {
        STDERR("streamWait");
        exit(1);
    }
    if (!cer1) {
        STDERR("cer1");
        exit(1);        
    }
    if (!ces1) {
        STDERR("ces1");
        exit(1);        
    }
    if (!cer2) {
        STDERR("cer2");
        exit(1);        
    }
    if (!ces2) {
        STDERR("ces2");
        exit(1);       
    }

    std::vector<std::shared_ptr<CpuNode>> order1, order2;
    order1.push_back(start);
    order1.push_back(postRecv);
    order1.push_back(yl);
    order1.push_back(streamWait);
    order1.push_back(scatter);
    order1.push_back(cer1);
    order1.push_back(ces1);
    order1.push_back(postSend);
    order1.push_back(waitRecv);
    order1.push_back(yr);
    order1.push_back(waitSend);
    order1.push_back(y);
    order1.push_back(cer2);
    order1.push_back(ces2);
    order1.push_back(end);

    order2.push_back(start);
    order2.push_back(scatter);
    order2.push_back(cer1);
    order2.push_back(ces1);
    order2.push_back(postRecv);
    order2.push_back(postSend);
    order2.push_back(waitRecv);
    order2.push_back(yr);
    order2.push_back(waitSend);
    order2.push_back(yl);
    order2.push_back(streamWait);
    order2.push_back(y);
    order2.push_back(cer2);
    order2.push_back(ces2);
    order2.push_back(end);

    BenchOpts opts;
    opts.nIters = 200;
    #if 0
    double *p;
    hipMalloc(&p, 8 * 1000000ull);

    {
        double last;
        double current = std::numeric_limits<double>::infinity();
        do {
            last = current;
            double start = MPI_Wtime();
            MPI_Barrier(MPI_COMM_WORLD);
            for (int i = 0; i < 100; ++i) {
                MPI_Allreduce(MPI_IN_PLACE, p, 1000000ull, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
            }
            current = MPI_Wtime() - start;
            if (0 == rank) {
                std::cout << current << "\n";
            }
            
        } while (current < last);
    }

    for (int i = 0; i < 100; ++i) {
    MPI_Allreduce(MPI_IN_PLACE, p, 1024, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    }


    for (int i = 0; i < 0; ++i) {
        Benchmark::Result br =
        benchmarker.benchmark(order1, MPI_COMM_WORLD, opts);
        
        if (0 == rank) {
            std::cout << "order1 "
            << "01=" << br.pct01
            << " 10=" << br.pct10
            << " 50=" << br.pct50
            << " 90=" << br.pct90
            << " 99=" << br.pct99
            << " st=" << br.stddev
            << "\n"
            ;
        }
    }

    {
        double last;
        double current = std::numeric_limits<double>::infinity();
        do {
            last = current;
            Benchmark::Result br =
            benchmarker.benchmark(order2, MPI_COMM_WORLD, opts);
            current = br.pct10;
            if (0 == rank) {
                std::cout << current << "\n";
            }
            
        } while (current < last);
    }
#endif
    for (int i = 0; i < 10; ++i) {
        Benchmark::Result br =
        benchmarker.benchmark(order2, MPI_COMM_WORLD, opts);
        
        if (0 == rank) {
            std::cout << "order2 "
            << "01=" << br.pct01
            << " 10=" << br.pct10
            << " 50=" << br.pct50
            << " 90=" << br.pct90
            << " 99=" << br.pct99
            << " st=" << br.stddev
            << "\n"
            << std::flush;
        }
    }

#if 1
    for (int i = 0; i < 10; ++i) {
        Benchmark::Result br =
        benchmarker.benchmark(order1, MPI_COMM_WORLD, opts);
        
        if (0 == rank) {
            std::cout << "order1 "
            << "01=" << br.pct01
            << " 10=" << br.pct10
            << " 50=" << br.pct50
            << " 90=" << br.pct90
            << " 99=" << br.pct99
            << " st=" << br.stddev
            << "\n"
            << std::flush;
        }
    }

    for (int i = 0; i < 10; ++i) {
        Benchmark::Result br =
        benchmarker.benchmark(order2, MPI_COMM_WORLD, opts);
        
        if (0 == rank) {
            std::cout << "order2 "
            << "01=" << br.pct01
            << " 10=" << br.pct10
            << " 50=" << br.pct50
            << " 90=" << br.pct90
            << " 99=" << br.pct99
            << " st=" << br.stddev
            << "\n"
            << std::flush;
        }
    }

    for (int i = 0; i < 10; ++i) {
        Benchmark::Result br =
        benchmarker.benchmark(order1, MPI_COMM_WORLD, opts);
        
        if (0 == rank) {
            std::cout << "order1 "
            << "01=" << br.pct01
            << " 10=" << br.pct10
            << " 50=" << br.pct50
            << " 90=" << br.pct90
            << " 99=" << br.pct99
            << " st=" << br.stddev
            << "\n"
            << std::flush;
        }
        br =
        benchmarker.benchmark(order2, MPI_COMM_WORLD, opts);
        
        if (0 == rank) {
            std::cout << "order2 "
            << "01=" << br.pct01
            << " 10=" << br.pct10
            << " 50=" << br.pct50
            << " 90=" << br.pct90
            << " 99=" << br.pct99
            << " st=" << br.stddev
            << "\n"
            << std::flush;
        }
    }
#endif
    return 0;
}

int main(int argc, char **argv)
{

    MPI_Init(&argc, &argv);


    if (argc > 1) {
        CsvBenchmarker b(argv[1]);
        return do_comparison(b, "");
    } else {
        EmpiricalBenchmarker b;
        return do_comparison(b, "");
    }


    MPI_Finalize();

    return 0;
}