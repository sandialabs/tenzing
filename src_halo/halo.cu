#include "sched/numeric.hpp"
#include "sched/operation.hpp"
#include "sched/schedule.hpp"

#include "ops_halo_exchange.hpp"

#include <mpi.h>

#include <vector>
#include <memory>
#include <algorithm>

int main(int argc, char **argv) {

    typedef HaloExchange::StorageOrder StorageOrder;
    typedef HaloExchange::Args Args;

    MPI_Init(&argc, &argv);

    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    typedef double Real;

    hipStream_t stream1, stream2;
    CUDA_RUNTIME(hipStreamCreate(&stream1));
    CUDA_RUNTIME(hipStreamCreate(&stream2));
    if (stream1 > stream2) std::swap(stream1, stream2);

    
    Args args;
    args.nQ = 3; // quantities per gridpoint
    args.nX = 128; // x and y extent of cells / rank
    args.nY = 128;
    args.nZ = 128;
    args.pitch = 512; // pitch of allocated memory in bytes
    args.nGhost = 3; // ghost cell radius
    args.storageOrder = StorageOrder::QXYZ;


    /* allocate width * height * depth
    */
    {
        size_t pitch, d2, d3, d4;
        switch(args.storageOrder) {
            case StorageOrder::QXYZ: {
                pitch = (sizeof(double) * args.nQ + args.pitch - 1) / args.pitch * args.pitch;
                d2 = args.nX + 2 * args.nGhost;
                d3 = args.nY + 2 * args.nGhost;
                d4 = args.nZ + 2 * args.nGhost;
                break;
            }
            default:
            THROW_RUNTIME("unhandled storage order");
        }

        std::cerr << "alloc p= " << pitch << " d2=" << d2 << " d3=" << d3 << " d4=" << d4 
                  << " (" << pitch * d2 * d3 * d4 / 1024.0 / 1024.0 << "MiB)\n";
        CUDA_RUNTIME(hipMalloc(&args.grid, pitch * d2 * d3 * d4));
    }

    // rank dimensions
    Dim3<int64_t> rd(1,1,1);

    {
        for (const auto &pf : prime_factors(size)) {
            if (rd.x < rd.y && rd.x < rd.z) {
                rd.x *= pf;
            } else if (rd.y < rd.z) {
                rd.y *= pf;
            } else {
                rd.z *= pf;
            }
        }
        if (0 == rank) std::cerr << "rank grid: " << rd << "\n";

    }

    if (size != rd.x * rd.y * rd.z) {
        THROW_RUNTIME("size " << size << " did not match rank dims\n");
    }

    args.rankToCoord = [rd](int _rank) -> Dim3<int64_t> {
        Dim3<int64_t> coord;
        coord.x = _rank % rd.x;
        _rank /= rd.x;
        coord.y = _rank % rd.y;
        _rank /= rd.y;
        coord.z = _rank % rd.z;
        return coord;
    };
    args.coordToRank = [size, rd](const Dim3<int64_t> &coord) -> int {

        Dim3<int64_t> wrapped(coord);

        // wrap out of bounds
        while(wrapped.x < 0) {
            wrapped.x += rd.x;
        }
        while(wrapped.y < 0) {
            wrapped.y += rd.y;
        }
        while(wrapped.z < 0) {
            wrapped.z += rd.z;
        }
        wrapped.x = wrapped.x % rd.x;
        wrapped.y = wrapped.y % rd.y;
        wrapped.z = wrapped.z % rd.z;

        int _rank = wrapped.x + wrapped.y * rd.x + wrapped.z * rd.x * rd.y;
        if (_rank >= size || _rank < 0) {
            THROW_RUNTIME("invalid computed rank " << _rank);
        }
        return _rank;
    };


    std::cerr << "create nodes\n";
    std::shared_ptr<Start> start = std::make_shared<Start>();
    std::shared_ptr<HaloExchange> exchange = std::make_shared<HaloExchange>(args);
    std::shared_ptr<End> end = std::make_shared<End>();

    std::cerr << "create graph\n";
    Graph<Node> orig(start);
    orig.then(start, exchange);
    orig.then(exchange, end);

    if (0 == rank) {
        orig.dump_graphviz("orig.dot");
    }

    std::cerr << "expand\n";
    exchange->expand_in(orig);

    if (0 == rank) {
        std::cerr << "dump\n";
        orig.dump_graphviz("expanded.dot");
    }

    std::cerr << "assign streams\n";
    std::vector<Graph<Node>> gpuGraphs = use_streams2(orig, {stream1, stream2});

    if (0 == rank) {
        std::cerr << "dump\n";
        gpuGraphs[0].dump_graphviz("gpu_0.dot");
    }

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "insert sync...\n";
    std::vector<Graph<Node>> syncedGraphs;
    for (auto &graph : gpuGraphs) {
        auto next = insert_synchronization(graph);
        syncedGraphs.push_back(next);
    }

    if (0 == rank) {
        std::cerr << "dump\n";
        syncedGraphs[0].dump_graphviz("sync_0.dot");
        syncedGraphs[5].dump_graphviz("sync_5.dot");
    }


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "convert to cpu graphs...\n";
    std::vector<Graph<CpuNode>> cpuGraphs;
    for (auto &graph : syncedGraphs) {
        cpuGraphs.push_back(graph.nodes_cast<CpuNode>());
    }
    if (0 == rank) std::cerr << "converted " << cpuGraphs.size() << " graphs\n";

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "create orderings...\n";
    std::vector<Schedule> schedules;
    for (auto &graph : cpuGraphs) {
        auto ss = make_schedules_random(graph, 10);
        for (auto &s : ss) {
            schedules.push_back(s);
        }
        std::cerr << ".";
    }
    std::cerr << "\n";
    std::cerr << "created " << schedules.size() << " schedules\n";

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "remove redundant syncs schedules...\n";
    for (auto &sched : schedules) {
        int count = sched.remove_redundant_syncs();
    }

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "sort schedules...\n";
    std::sort(schedules.begin(), schedules.end(), Schedule::by_node_typeid);


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "eliminate equivalent schedules...\n";
    {
        int count = 0;
        size_t total = schedules.size() * (schedules.size() - 1);
        int next = 99;
        for (size_t i = 0; i < schedules.size(); ++i) {
            for (size_t j = i+1; j < schedules.size(); ++j) {
                if (Schedule::predicate(schedules[i], schedules[j])) {
                    schedules.erase(schedules.begin() + j);
                    count += 1;
                    --j; // since we need to check the schedule that is now in j
                }
                size_t left = (schedules.size() - i) * (schedules.size() - i - 1);
                if (left < next * total / 100) {
                    if (0 == rank) std::cerr << next << "% (~" << (schedules.size()-i) * (schedules.size() - i - 1) << " comparisons left...)\n";
                    next = left * 100 / total;
                }
            }
        }
        std::cerr << "found " << count << " duplicate schedules\n";
        std::cerr << "found " << schedules.size() << " unique schedules\n";
    }

    if (0 == rank) std::cerr << "testing schedules...\n";
    for (size_t i = 0; i < schedules.size(); ++i) {
    // for (size_t i = 53; i < 54; ++i) {
        if (0 == rank) std::cerr << " " << i;
        MPI_Barrier(MPI_COMM_WORLD);
        schedules[i].run();
        MPI_Barrier(MPI_COMM_WORLD);
    }
    if (0 == rank) std::cerr << std::endl;
    if (0 == rank) std::cerr << "done" << std::endl;


    if (0 == rank) std::cerr << "benching schedules...\n";
    Schedule::BenchOpts opts;
    opts.nIters = 100;
    auto benchResults = Schedule::benchmark(schedules, MPI_COMM_WORLD, opts);
    if (0 == rank) std::cerr << "done" << std::endl;

    if (0 == rank)
    {
        std::cout << "1pctl,10pctl,50pctl,90pct,99pct,stddev,order\n";
        for (size_t si = 0; si < benchResults.size(); ++si) {
            auto &result = benchResults[si];
            std::cout 
                   << result.pct01 
            << "," << result.pct10 
            << "," << result.pct50 
            << "," << result.pct90 
            << "," << result.pct99 
            << "," << result.stddev;

            for (auto &op : schedules[si].order) {
                std::cout << "," << op->name();
            }

            std::cout << "\n";
        }
    }

    MPI_Finalize();
    return 0;
}