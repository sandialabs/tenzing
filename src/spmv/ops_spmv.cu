/* Copyright 2022 National Technology & Engineering Solutions of Sandia, LLC (NTESS). Under the
 * terms of Contract DE-NA0003525 with NTESS, the U.S. Government retains certain rights in this
 * software.
 */


#include "tenzing/spmv/ops_spmv.cuh"

void PostSend::run(Platform & /*plat*/) {
  for (Isend::Args &args : args_.sends) {
  #ifdef TENZING_RUNTIME_SANITY_CHECKS
    if (!args.buf) THROW_RUNTIME("bad buf");
    if (!args.request) THROW_RUNTIME("bad request");
    {
      hipPointerAttribute_t attrs;
      CUDA_RUNTIME(hipPointerGetAttributes(&attrs, args.buf));
      if (attrs.type == cudaMemoryTypeUnregistered) {
        THROW_RUNTIME("postsend buffer is unregistered");
      }
    }
  #endif
    MPI_Isend(args.buf, args.count, args.datatype, args.dest, args.tag, args.comm, args.request);
  }
}

void Scatter::run(hipStream_t stream) {
  #ifdef TENZING_RUNTIME_SANITY_CHECKS
      if (args_.dst.size() != args_.idx.size()) {
        THROW_RUNTIME("scatter dst size was different than index size");
      }
      if (!args_.dst.data_) {
          THROW_RUNTIME("bad dst");
      }
      if (!args_.src.data_) {
          THROW_RUNTIME("bad src");
      }
      if (!args_.idx.data_) {
          THROW_RUNTIME("bad idx");
      }
  #endif
      scatter<<<128, 100, 0, stream>>>(args_.dst, args_.src, args_.idx);
      CUDA_RUNTIME(hipGetLastError());
    }

void VectorAdd::run(hipStream_t /*stream*/) {
  #warning VectorAdd::run(hipStream_t) is a no-op
};
