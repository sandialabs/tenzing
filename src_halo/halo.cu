#include "sched/operation.hpp"
#include "sched/schedule.hpp"

#include "ops_halo_exchange.hpp"

#include <mpi.h>

#include <vector>
#include <memory>
#include <algorithm>

int main(int argc, char **argv) {

    typedef HaloExchange::StorageOrder StorageOrder;
    typedef HaloExchange::Args Args;

    MPI_Init(&argc, &argv);

    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    typedef double Real;

    hipStream_t stream1, stream2;
    CUDA_RUNTIME(hipStreamCreate(&stream1));
    CUDA_RUNTIME(hipStreamCreate(&stream2));
    if (stream1 > stream2) std::swap(stream1, stream2);

    
    Args args;
    args.nQ = 3; // quantities per gridpoint
    args.nX = 128; // x and y extent of cells / rank
    args.nY = 128;
    args.pitch = 512; // pitch of allocated memory in bytes
    args.nGhost = 3; // ghost cell radius
    args.storageOrder = StorageOrder::QXY;


    /* allocate width * height * depth
    */
    std::vector<Real> hostGrid;
    {
        size_t width, height, depth;
        switch(args.storageOrder) {
            case StorageOrder::QXY: {
                width = (args.nX + args.pitch - 1) / args.pitch * args.pitch;
                height = args.nX + 2 * args.nGhost;
                depth = args.nY + 2 * args.nGhost;
            }
        }

        hostGrid.resize(width * height * depth);
        CUDA_RUNTIME(hipMalloc(&args.grid, width * height * depth));
    }

    // rank dimensions
    Dim2<int64_t> rd(1,1);

    if (size != rd.x * rd.y) {
        THROW_RUNTIME("size " << size << " did not match rank dims\n");
    }

    args.rankToCoord = [rd](int _rank) -> Dim2<int64_t> {
        Dim2<int64_t> coord;
        coord.x = _rank % rd.x;
        coord.y = _rank / rd.x;
        return coord;
    };
    args.coordToRank = [size, rd](const Dim2<int64_t> &coord) -> int {

        Dim2<int64_t> wrapped(coord);

        // wrap out of bounds
        while(wrapped.x < 0) {
            wrapped.x += rd.x;
        }
        while(wrapped.y < 0) {
            wrapped.y += rd.y;
        }
        wrapped.x = wrapped.x % rd.x;
        wrapped.y = wrapped.y % rd.y;

        int _rank = wrapped.x + wrapped.y * rd.x;
        if (_rank >= size || _rank < 0) {
            THROW_RUNTIME("invalid computed rank " << _rank);
        }
        return _rank;
    };


    std::cerr << "create nodes\n";
    std::shared_ptr<Start> start = std::make_shared<Start>();
    std::shared_ptr<HaloExchange> exchange = std::make_shared<HaloExchange>(args);
    std::shared_ptr<End> end = std::make_shared<End>();

    std::cerr << "create graph\n";
    Graph<Node> orig(start);
    orig.then(start, exchange);
    orig.then(exchange, end);

    if (0 == rank) {
        orig.dump_graphviz("orig.dot");
    }

    std::cerr << "expand\n";
    exchange->expand_in(orig);

    if (0 == rank) {
        std::cerr << "dump\n";
        orig.dump_graphviz("expanded.dot");
    }

    std::cerr << "assign streams\n";
    std::vector<Graph<Node>> gpuGraphs = use_streams2(orig, {stream1, stream2});

    if (0 == rank) {
        std::cerr << "dump\n";
        gpuGraphs[0].dump_graphviz("gpu_0.dot");
    }

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "insert sync...\n";
    std::vector<Graph<Node>> syncedGraphs;
    for (auto &graph : gpuGraphs) {
        auto next = insert_synchronization(graph);
        syncedGraphs.push_back(next);
    }

    if (0 == rank) {
        std::cerr << "dump\n";
        syncedGraphs[0].dump_graphviz("sync_0.dot");
    }


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "convert to cpu graphs...\n";
    std::vector<Graph<CpuNode>> cpuGraphs;
    for (auto &graph : syncedGraphs) {
        cpuGraphs.push_back(graph.nodes_cast<CpuNode>());
    }
    if (0 == rank) std::cerr << "converted " << cpuGraphs.size() << " graphs\n";

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "create orderings...\n";
    std::vector<Schedule> schedules;
    for (auto &graph : cpuGraphs) {
        auto ss = make_schedules_random(graph, 10);
        for (auto &s : ss) {
            schedules.push_back(s);
        }
    }
    std::cerr << "created " << schedules.size() << " schedules\n";

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "sort schedules...\n";
    std::sort(schedules.begin(), schedules.end(), Schedule::by_node_typeid);


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "eliminate equivalent schedules...\n";
    {
        int count = 0;
        size_t total = schedules.size() * (schedules.size() - 1);
        int next = 99;
        for (size_t i = 0; i < schedules.size(); ++i) {
            for (size_t j = i+1; j < schedules.size(); ++j) {
                if (Schedule::predicate(schedules[i], schedules[j])) {
                    schedules.erase(schedules.begin() + j);
                    count += 1;
                    --j; // since we need to check the schedule that is now in j
                }
                size_t left = (schedules.size() - i) * (schedules.size() - i - 1);
                if (left < next * total / 100) {
                    if (0 == rank) std::cerr << next << "% (~" << (schedules.size()-i) * (schedules.size() - i - 1) << " comparisons left...)\n";
                    next = left * 100 / total;
                }
            }
        }
        std::cerr << "found " << count << " duplicate schedules\n";
        std::cerr << "found " << schedules.size() << " unique schedules\n";
    }

    if (0 == rank) std::cerr << "testing schedules...\n";
    for (size_t i = 0; i < schedules.size(); ++i) {
    // for (size_t i = 9; i < 10; ++i) {
        if (0 == rank) std::cerr << " " << i;
        MPI_Barrier(MPI_COMM_WORLD);
        schedules[i].run();
        MPI_Barrier(MPI_COMM_WORLD);
    }
    if (0 == rank) std::cerr << std::endl;
    if (0 == rank) std::cerr << "done" << std::endl;



    MPI_Finalize();
    return 0;
}