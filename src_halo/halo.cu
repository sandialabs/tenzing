#include "sched/numeric.hpp"
#include "sched/operation.hpp"
#include "sched/schedule.hpp"
#include "sched/benchmarker.hpp"

#include "ops_halo_exchange.hpp"

#include <mpi.h>

#include <vector>
#include <memory>
#include <algorithm>

int main(int argc, char **argv) {

    typedef HaloExchange::StorageOrder StorageOrder;
    typedef HaloExchange::Args Args;

    MPI_Init(&argc, &argv);

    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    typedef double Real;

    Platform plat = Platform::make_n_streams(2, MPI_COMM_WORLD);
    
    Args args;
    args.nQ = 3; // quantities per gridpoint
    args.nX = 512; // x and y extent of cells / rank
    args.nY = 512;
    args.nZ = 512;
    args.pitch = 128; // pitch of allocated memory in bytes
    args.nGhost = 3; // ghost cell radius
    args.storageOrder = StorageOrder::XYZQ;


    /* allocate width * height * depth
    */
    {
        size_t pitch, d2, d3, d4;
        switch(args.storageOrder) {
            case StorageOrder::QXYZ: {
                pitch = (sizeof(double) * args.nQ + args.pitch - 1) / args.pitch * args.pitch;
                d2 = args.nX + 2 * args.nGhost;
                d3 = args.nY + 2 * args.nGhost;
                d4 = args.nZ + 2 * args.nGhost;
                break;
            }
            case StorageOrder::XYZQ: {
                pitch = round_up(sizeof(double) * (args.nX + 2 * args.nGhost), args.pitch);
                d2 = args.nY + 2 * args.nGhost;
                d3 = args.nZ + 2 * args.nGhost;
                d4 = args.nQ;
                break;
            }
            default:
            THROW_RUNTIME("unhandled storage order");
        }

        std::cerr << "alloc p=" << pitch << " d2=" << d2 << " d3=" << d3 << " d4=" << d4 
                  << " (" << pitch * d2 * d3 * d4 / 1024.0 / 1024.0 << "MiB)\n";
        CUDA_RUNTIME(hipMalloc(&args.grid, pitch * d2 * d3 * d4));
    }

    // rank dimensions
    Dim3<int64_t> rd(1,1,1);

    {
        for (const auto &pf : prime_factors(size)) {
            if (rd.x < rd.y && rd.x < rd.z) {
                rd.x *= pf;
            } else if (rd.y < rd.z) {
                rd.y *= pf;
            } else {
                rd.z *= pf;
            }
        }
        if (0 == rank) std::cerr << "rank grid: " << rd << "\n";

    }

    if (size != rd.x * rd.y * rd.z) {
        THROW_RUNTIME("size " << size << " did not match rank dims\n");
    }

    args.rankToCoord = [rd](int _rank) -> Dim3<int64_t> {
        Dim3<int64_t> coord;
        coord.x = _rank % rd.x;
        _rank /= rd.x;
        coord.y = _rank % rd.y;
        _rank /= rd.y;
        coord.z = _rank % rd.z;
        return coord;
    };
    args.coordToRank = [size, rd](const Dim3<int64_t> &coord) -> int {

        Dim3<int64_t> wrapped(coord);

        // wrap out of bounds
        while(wrapped.x < 0) {
            wrapped.x += rd.x;
        }
        while(wrapped.y < 0) {
            wrapped.y += rd.y;
        }
        while(wrapped.z < 0) {
            wrapped.z += rd.z;
        }
        wrapped.x = wrapped.x % rd.x;
        wrapped.y = wrapped.y % rd.y;
        wrapped.z = wrapped.z % rd.z;

        int _rank = wrapped.x + wrapped.y * rd.x + wrapped.z * rd.x * rd.y;
        if (_rank >= size || _rank < 0) {
            THROW_RUNTIME("invalid computed rank " << _rank);
        }
        return _rank;
    };


    std::cerr << "create nodes\n";
    std::shared_ptr<Start> start = std::make_shared<Start>();
    std::shared_ptr<HaloExchange> exchange = std::make_shared<HaloExchange>(args);
    std::shared_ptr<End> end = std::make_shared<End>();

    std::cerr << "create graph\n";
    Graph<OpBase> orig(start);
    orig.then(start, exchange);
    orig.then(exchange, end);

    if (0 == rank) {
        orig.dump_graphviz("orig.dot");
    }

#if 0
    std::cerr << "expand\n";
    exchange->expand_in(orig);

    if (0 == rank) {
        std::cerr << "dump\n";
        orig.dump_graphviz("expanded.dot");
    }

    std::cerr << "assign streams\n";
    std::vector<Graph<OpBase>> gpuGraphs = use_streams2(orig, {stream1, stream2});

    if (0 == rank) {
        std::cerr << "dump\n";
        gpuGraphs[0].dump_graphviz("gpu_0.dot");
    }


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "insert sync...\n";
    std::vector<Graph<OpBase>> syncedGraphs;
    for (auto &graph : gpuGraphs) {
        auto next = insert_synchronization(graph);
        syncedGraphs.push_back(next);
    }

    if (0 == rank) {
        std::cerr << "dump\n";
        syncedGraphs[0].dump_graphviz("sync_0.dot");
        syncedGraphs[5].dump_graphviz("sync_5.dot");
    }
#else
    std::cerr << "expand and assign streams\n";
    exchange->expand_3d_streams(orig, stream1, stream2, stream2);

    if (0 == rank) {
        std::cerr << "dump\n";
        orig.dump_graphviz("expanded.dot");
    }

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "insert sync...\n";
    std::vector<Graph<OpBase>> syncedGraphs;
    syncedGraphs.push_back(insert_synchronization(orig));

    if (0 == rank) {
        std::cerr << "dump\n";
        syncedGraphs[0].dump_graphviz("sync_0.dot");
    }
#endif

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "convert to cpu graphs...\n";
    std::vector<Graph<CpuOp>> cpuGraphs;
    for (auto &graph : syncedGraphs) {
        cpuGraphs.push_back(graph.nodes_cast<CpuOp>());
    }
    if (0 == rank) std::cerr << "converted " << cpuGraphs.size() << " graphs\n";

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "create orderings...\n";
    std::vector<Schedule> schedules;
    for (auto &graph : cpuGraphs) {
        auto ss = make_schedules_random(graph, 1000);
        for (auto &s : ss) {
            schedules.push_back(s);
        }
        std::cerr << ".";
    }
    std::cerr << "\n";
    std::cerr << "created " << schedules.size() << " schedules\n";

#if 0
    MPI_Barrier(MPI_COMM_WORLD);
    for (size_t si = 10000; si < 10010; ++si) {
        for (auto &op : schedules[si].order) {
            if (0 == rank) std::cerr << "," << op->name();
        }
        if (0 == rank) std::cerr << "\n";
    }
#endif

    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "remove redundant syncs schedules...\n";
    for (auto &sched : schedules) {
        int count = sched.remove_redundant_syncs();
        if (0 == rank) std::cerr << count << " ";
    }
    if (0 == rank) std::cerr << "\n";

#if 0
    MPI_Barrier(MPI_COMM_WORLD);
    for (size_t si = 10000; si < 10010; ++si) {
        for (auto &op : schedules[si].order) {
            if (0 == rank) std::cerr << "," << op->name();
        }
        if (0 == rank) std::cerr << "\n";
    }
#endif


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "sort schedules...\n";
    std::sort(schedules.begin(), schedules.end(), Schedule::by_op_typeid);


    MPI_Barrier(MPI_COMM_WORLD);
    if (0 == rank) std::cerr << "eliminate equivalent schedules...\n";
    {
        int count = 0;
        size_t total = schedules.size() * (schedules.size() - 1);
        int next = 99;
        for (size_t i = 0; i < schedules.size(); ++i) {
            for (size_t j = i+1; j < schedules.size(); ++j) {
                if (Schedule::predicate(schedules[i], schedules[j])) {
                    schedules.erase(schedules.begin() + j);
                    count += 1;
                    --j; // since we need to check the schedule that is now in j
                }
                size_t left = (schedules.size() - i) * (schedules.size() - i - 1);
                if (left < next * total / 100) {
                    if (0 == rank) std::cerr << next << "% (~" << (schedules.size()-i) * (schedules.size() - i - 1) << " comparisons left...)\n";
                    next = left * 100 / total;
                }
            }
        }
        std::cerr << "found " << count << " duplicate schedules\n";
        std::cerr << "found " << schedules.size() << " unique schedules\n";
    }

    MPI_Barrier(MPI_COMM_WORLD);





    if (0 == rank) std::cerr << "testing schedules...\n";
    for (size_t i = 0; i < schedules.size(); ++i) {
    // for (size_t i = 53; i < 54; ++i) {
        if (0 == rank) std::cerr << " " << i;
        MPI_Barrier(MPI_COMM_WORLD);
        schedules[i].run();
        MPI_Barrier(MPI_COMM_WORLD);
    }
    if (0 == rank) std::cerr << std::endl;
    if (0 == rank) std::cerr << "done" << std::endl;


    if (0 == rank) std::cerr << "benching schedules...\n";
    BenchOpts opts;
    opts.nIters = 100;
    EmpiricalBenchmarker benchmarker;
    auto benchResults = benchmarker.benchmark(schedules, MPI_COMM_WORLD, opts);
    if (0 == rank) std::cerr << "done" << std::endl;

    if (0 == rank)
    {
        std::cout << "1pctl,10pctl,50pctl,90pct,99pct,stddev,order\n";
        for (size_t si = 0; si < benchResults.size(); ++si) {
            auto &result = benchResults[si];
            std::cout 
                   << result.pct01 
            << "," << result.pct10 
            << "," << result.pct50 
            << "," << result.pct90 
            << "," << result.pct99 
            << "," << result.stddev;

            for (auto &op : schedules[si].order) {
                std::cout << "," << op->name();
            }

            std::cout << "\n";
        }
    }

    MPI_Finalize();
    return 0;
}